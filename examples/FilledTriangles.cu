#include "hip/hip_runtime.h"
/*
 #
 #  File        : CImg_demo.cpp
 #                ( C++ source file )
 #
 #  Description : A multi-part demo demonstrating some of the CImg capabilities.
 #                This file is a part of the CImg Library project.
 #                ( http://cimg.sourceforge.net )
 #
 #  Copyright   : David Tschumperle
 #                ( http://tschumperle.users.greyc.fr/ )
 #
 #  License     : CeCILL v2.0
 #                ( http://www.cecill.info/licences/Licence_CeCILL_V2-en.html )
 #
 #  This software is governed by the CeCILL  license under French law and
 #  abiding by the rules of distribution of free software.  You can  use,
 #  modify and/ or redistribute the software under the terms of the CeCILL
 #  license as circulated by CEA, CNRS and INRIA at the following URL
 #  "http://www.cecill.info".
 #
 #  As a counterpart to the access to the source code and rights to copy,
 #  modify and redistribute granted by the license, users are provided only
 #  with a limited warranty  and the software's author,  the holder of the
 #  economic rights,  and the successive licensors  have only  limited
 #  liability.
 #
 #  In this respect, the user's attention is drawn to the risks associated
 #  with loading,  using,  modifying and/or developing or reproducing the
 #  software by the user in light of its specific status of free software,
 #  that may mean  that it is complicated to manipulate,  and  that  also
 #  therefore means  that it is reserved for developers  and  experienced
 #  professionals having in-depth computer knowledge. Users are therefore
 #  encouraged to load and test the software's suitability as regards their
 #  requirements in conditions enabling the security of their systems and/or
 #  data to be ensured and,  more generally, to use and operate it in the
 #  same conditions as regards security.
 #
 #  The fact that you are presently reading this means that you have had
 #  knowledge of the CeCILL license and that you accept its terms.
 #
*/

// Include static image data, so that the exe does not depend on external image files.
#include "img/CImg_demo.h"
#include <iostream>
#include <iomanip>
//Include the nVidia CUDA runtime for Parallel programming

#include <hiprand/hiprand_kernel.h>

// Include CImg library header.
#include "CImg.h"
using namespace cimg_library;
#undef min
#undef max

/*
 * Setup and initialize hiprand with a seed
 */
__global__ void initCurand(hiprandState* state){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(100, idx, 0, &state[idx]);
	__syncthreads();
}

/*
 * CUDA kernel that will execute 100 threads in parallel
*/

__global__ void initializeArrays(float* posx, float* posy,float* rayon, float* veloc, float* opacity
								,float* angle, unsigned char* color, int height, int width, hiprandState* state, size_t pitch){

	int idx =  blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState localState = state[idx];

	posx[idx] = (float)(hiprand_normal(&localState)*width);
    posy[idx] = (float)(hiprand_normal(&localState)*height);
    rayon[idx] = (float)(10 + hiprand_normal(&localState)*50);
    angle[idx] = (float)(hiprand_normal(&localState)*360);
    veloc[idx] = (float)(hiprand_uniform(&localState)*20 - 10);
    color[idx*pitch] = (unsigned char)(hiprand_normal(&localState)*255);
    color[(idx*pitch)+1] = (unsigned char)(hiprand_normal(&localState)*255);
    color[(idx*pitch)+2] = (unsigned char)(hiprand_normal(&localState)*255);
    opacity[idx] = (float)(0.3f + 1.5f *hiprand_normal(&localState));
	
	__syncthreads();
}

void errCheck(hipError_t err, const char* msg){
	 if (err != hipSuccess)
        std::cout<< msg << ": " << hipGetErrorString(err) << std::endl;
}

/*---------------------------

  Main procedure

  --------------------------*/
int main() {

    // Create a colored 640x480 background image which consists of different color shades.
    CImg<float> background(640,480,1,3);
    cimg_forXY(background,x,y) background.fillC(x,y,0,
                                                x*std::cos(6.0*y/background.height()) + y*std::sin(9.0*x/background.width()),
                                                x*std::sin(8.0*y/background.height()) - y*std::cos(11.0*x/background.width()),
                                                x*std::cos(13.0*y/background.height()) - y*std::sin(8.0*x/background.width()));
    background.normalize(0,180);
    
    // Init images and create display window.
    CImg<unsigned char> img0(background), img;
    unsigned char white[] = { 255, 255, 255 }, color[100][3];
    CImgDisplay disp(img0,"[#6] - Filled Triangles (Click to shrink)");

    // Define random properties (pos, size, colors, ..) for all triangles that will be displayed.
    float posx[100];
	float posy[100];
	float rayon[100];
	float angle[100];
	float veloc[100];
	float opacity[100];
	// Define the same properties but for the device
	float* d_posx;
	float* d_posy;
	float* d_rayon;
	float* d_angle;
	float* d_veloc;
	float* d_opacity;
	unsigned char* d_color;

	// CURAND state
	hiprandState* devState;
	// error handling
	hipError_t err;

	// allocate memory on the device for the device arrays
	err = hipMalloc((void**)&d_posx, 100 * sizeof(float));
	errCheck(err, "hipMalloc((void**)&d_posx, 100 * sizeof(float))");
	err = hipMalloc((void**)&d_posy, 100 * sizeof(float));
	errCheck(err,"hipMalloc((void**)&d_posy, 100 * sizeof(float))");
	err = hipMalloc((void**)&d_rayon, 100 * sizeof(float));
	errCheck(err,"hipMalloc((void**)&d_rayon, 100 * sizeof(float))");
    err = hipMalloc((void**)&d_angle, 100 * sizeof(float));
	errCheck(err,"hipMalloc((void**)&d_angle, 100 * sizeof(float))");
	err = hipMalloc((void**)&d_veloc, 100 * sizeof(float));
	errCheck(err,"hipMalloc((void**)&d_veloc, 100 * sizeof(float))");
	err = hipMalloc((void**)&d_opacity, 100 * sizeof(float));
	errCheck(err,"hipMalloc((void**)&d_opacity, 100 * sizeof(float))");
	err = hipMalloc((void**)&devState, 100*sizeof(hiprandState));
	errCheck(err,"hipMalloc((void**)&devState, 100*sizeof(hiprandState))");
	size_t pitch;
	//allocated the device memory for source array  
	err = hipMallocPitch(&d_color, &pitch, 3 * sizeof(unsigned char),100);
	errCheck(err,"hipMallocPitch(&d_color, &pitch, 3 * sizeof(unsigned char),100)");
	// launch grid of threads
	dim3 dimBlock(100);
	dim3 dimGrid(1);
	  
	/* Kernel for initializing CURAND */
	initCurand<<<1,100>>>(devState);

	// synchronize the device and the host
    hipDeviceSynchronize();
     
	/*Kernel for initializing Arrays */
	initializeArrays<<<1, 100>>>(d_posx, d_posy, d_rayon, d_veloc, d_opacity, d_angle,
										d_color, img0.height(), img0.width(), devState, pitch);
	// synchronize the device and the host
    hipDeviceSynchronize();
	
	// get the populated arrays back to the host for use
	err = hipMemcpy(posx,d_posx, 100 * sizeof(float), hipMemcpyDeviceToHost);
	errCheck(err,"hipMemcpy(posx,d_posx, 100 * sizeof(float), hipMemcpyDeviceToHost)");
	err = hipMemcpy(posy,d_posy, 100 * sizeof(float), hipMemcpyDeviceToHost);
	errCheck(err,"hipMemcpy(posy,d_posy, 100 * sizeof(float), hipMemcpyDeviceToHost)");
	err = hipMemcpy(rayon,d_rayon, 100 * sizeof(float), hipMemcpyDeviceToHost);
	errCheck(err,"hipMemcpy(rayon,d_rayon, 100 * sizeof(float), hipMemcpyDeviceToHost)");
	err = hipMemcpy(veloc,d_veloc, 100 * sizeof(float), hipMemcpyDeviceToHost);
	errCheck(err,"hipMemcpy(veloc,d_veloc, 100 * sizeof(float), hipMemcpyDeviceToHost)");
	err = hipMemcpy(opacity,d_opacity, 100 * sizeof(float), hipMemcpyDeviceToHost);
	errCheck(err,"hipMemcpy(opacity,d_opacity, 100 * sizeof(float), hipMemcpyDeviceToHost)");
	err = hipMemcpy(angle,d_angle, 100 * sizeof(float), hipMemcpyDeviceToHost);
	errCheck(err,"hipMemcpy(angle,d_angle, 100 * sizeof(float), hipMemcpyDeviceToHost)");
	// pitch of color array is 3+1 padded
	err = hipMemcpy2D(color,4,d_color,pitch,3 *sizeof(unsigned char),3, hipMemcpyDeviceToHost);
	errCheck(err,"hipMemcpy2D(color,pitch,d_color,100*3,3 *sizeof(unsigned char),100* sizeof(unsigned char), hipMemcpyDeviceToHost)");
    // measuring time it takes for triangle animations in 1000 iterations
    int i = 0, num = 1;
    
    // Start animation loop.
    while (!disp.is_closed() && !disp.is_keyQ() && !disp.is_keyESC() && i < 1000) {
        img = img0;
        
        i++;
        // Draw each triangle on the background image.
        for (int k = 0; k<num; ++k) {
            const int
            x0 = (int)(posx[k] + rayon[k]*std::cos(angle[k]*cimg::PI/180)),
            y0 = (int)(posy[k] + rayon[k]*std::sin(angle[k]*cimg::PI/180)),
            x1 = (int)(posx[k] + rayon[k]*std::cos((angle[k] + 120)*cimg::PI/180)),
            y1 = (int)(posy[k] + rayon[k]*std::sin((angle[k] + 120)*cimg::PI/180)),
            x2 = (int)(posx[k] + rayon[k]*std::cos((angle[k] + 240)*cimg::PI/180)),
            y2 = (int)(posy[k] + rayon[k]*std::sin((angle[k] + 240)*cimg::PI/180));
            if (k%10) img.draw_triangle(x0,y0,x1,y1,x2,y2,color[k],opacity[k]);
            else img.draw_triangle(x0,y0,x1,y1,x2,y2,img0,0,0,img0.width()-1,0,0,img.height()-1,opacity[k]);
            img.draw_triangle(x0,y0,x1,y1,x2,y2,white,opacity[k],~0U);
            
            // Make the triangles rotate, and check for mouse click event.
            // (to make triangles collapse or join).
            angle[k]+=veloc[k];
            if (disp.mouse_x()>0 && disp.mouse_y()>0) {
                float u = disp.mouse_x() - posx[k], v = disp.mouse_y() - posy[k];
                if (disp.button()) { u = -u; v = -v; }
                posx[k]-=0.03f*u, posy[k]-=0.03f*v;
                if (posx[k]<0 || posx[k]>=img.width()) posx[k] = (float)(cimg::rand()*img.width());
                if (posy[k]<0 || posy[k]>=img.height()) posy[k] = (float)(cimg::rand()*img.height());
            }
        }
        
        // Display current animation framerate, and refresh display window.
        img.draw_text(5,5,"%u frames/s",white,0,0.5f,13,(unsigned int)disp.frames_per_second());
        img0.resize(disp.display(img).resize(false).wait(20));
        if (++num>100) num = 100;
        
        // Allow the user to toggle fullscreen mode, by pressing CTRL+F.
        if (disp.is_keyCTRLLEFT() && disp.is_keyF()) disp.resize(640,480,false).toggle_fullscreen(false);
    }

	// free allocated device memory
	hipFree(d_posy);
	hipFree(d_posx);
	hipFree(d_rayon);
	hipFree(d_veloc);
	hipFree(d_opacity);
	hipFree(d_color);
	hipFree(d_angle);
	hipFree(devState);
  return 0;
}